#include <iostream>
#include <fstream>
#include <vector>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

#define CHECK_CUDA(call) \
    { \
        const hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "Error: " << __FILE__ << ", line " << __LINE__ << ": " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    }

#define CHECK_CUSOLVER(call) \
    { \
        const hipsolverStatus_t status = call; \
        if (status != HIPSOLVER_STATUS_SUCCESS) { \
            std::cerr << "CUSOLVER error at line " << __LINE__ << std::endl; \
            exit(1); \
        } \
    }

void convertToColumnMajor(const std::vector<double> &rowMajor, std::vector<double> &colMajor, int n) {
    colMajor.resize(n * n);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            colMajor[j * n + i] = rowMajor[i * n + j];
        }
    }
}

void readMatrix(const std::string &filename, int &n, std::vector<double> &A, std::vector<double> &b, std::vector<double> &r) {
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(1);
    }

    infile >> n;

    A.resize(n * n);
    b.resize(n);
    r.resize(n);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            infile >> A[i * n + j];
        }
    }

    infile.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); // Skip empty line

    for (int i = 0; i < n; ++i) {
        infile >> b[i];
    }
    infile.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); // Skip empty line
    for (int i = 0; i < n; ++i) {
        infile >> r[i];
    }
    std::cout << std::endl;

    infile.close();
}

void solveGaussianElimination(int n, const std::vector<double> &A, const std::vector<double> &b, std::vector<double> &x) {
    double *d_A, *d_b;
    int *d_info;
    int *d_ipiv;
    hipsolverHandle_t cusolverH;
    hipStream_t stream;

    // Convert matrix to column-major order
    std::vector<double> A_colMajor;
    convertToColumnMajor(A, A_colMajor, n);
    double before = cpuSecond();

    CHECK_CUDA(hipMalloc((void **)&d_A, n * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void **)&d_b, n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void **)&d_info, sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_ipiv, n * sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_A, A_colMajor.data(), n * n * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b.data(), n * sizeof(double), hipMemcpyHostToDevice));

    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUSOLVER(hipsolverSetStream(cusolverH, stream));

    int workspace_size;
    double *d_workspace;

    CHECK_CUSOLVER(hipsolverDnDgetrf_bufferSize(cusolverH, n, n, d_A, n, &workspace_size));
    CHECK_CUDA(hipMalloc((void **)&d_workspace, workspace_size * sizeof(double)));

    // LU Factorization
    CHECK_CUSOLVER(hipsolverDnDgetrf(cusolverH, n, n, d_A, n, d_workspace, d_ipiv, d_info));

    // Check if the matrix is singular
    int info_host;
    CHECK_CUDA(hipMemcpy(&info_host, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info_host != 0) {
        std::cerr << "LU factorization failed, matrix is singular at U(" << info_host << "," << info_host << ")" << std::endl;
        hipFree(d_A);
        hipFree(d_b);
        hipFree(d_info);
        hipFree(d_ipiv);
        hipFree(d_workspace);
        hipsolverDnDestroy(cusolverH);
        hipStreamDestroy(stream);
        exit(1);
    }

    // Solve the linear system
    CHECK_CUSOLVER(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, n, 1, d_A, n, d_ipiv, d_b, n, d_info));

    // Copy the result back to the host
    x.resize(n);
    CHECK_CUDA(hipMemcpy(x.data(), d_b, n * sizeof(double), hipMemcpyDeviceToHost));

    // Cleanup
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_info);
    hipFree(d_ipiv);
    hipFree(d_workspace);
    hipsolverDnDestroy(cusolverH);
    hipStreamDestroy(stream);
    std::cout << "Time taken: " << cpuSecond() - before << " seconds" << std::endl;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file>" << std::endl;
        return 1;
    }

    int n;
    std::vector<double> A, b, x, r;
    readMatrix(argv[1], n, A, b, r);

    

    solveGaussianElimination(n, A, b, x);
    
    double largestError = -1;
    for (int i = 0; i < n; ++i) {
        double error = std::abs(x[i] - r[i]);
        if (error > largestError) {
            largestError = error;
        }
    }

    std::cout << "Largest error was " << largestError << std::endl;

    return 0;
}
